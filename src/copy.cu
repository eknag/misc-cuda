#include "hip/hip_runtime.h"
#include <assert.h>
#include <cstdint>
#include <cstdio>
#include <cuda/pipeline>
#include <cuda_pipeline_primitives.h>
#include <hip/hip_runtime.h>
#include <malloc.h>
#include <numeric>

#define gpuErrchk(ans)                                                         \
  {                                                                            \
    gpuAssert((ans), __FILE__, __LINE__);                                      \
  }
inline void gpuAssert(hipError_t code, const char *file, int line,
                      bool abort = true) {
  if (code != hipSuccess) {
    fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file,
            line);
    if (abort)
      exit(code);
  }
}

template <int ITEMS_PER_WARP, int THREADS>
__global__ void copy_and_measure(const int *__restrict__ A, int *__restrict__ B,
                                 int n, unsigned long long *times) {

  constexpr int total_items = ITEMS_PER_WARP;
  constexpr int items_per_thread = total_items / THREADS;
  constexpr int vector_size =
      4; // Using int4, which processes 4 integers at a time
  constexpr int vectors_per_thread = items_per_thread / vector_size;

  static_assert(total_items % (THREADS * vector_size) == 0,
                "ITEMS_PER_WARP must be divisible by THREADS * vector_size");

  int tid = threadIdx.x;

  __shared__ __align__(16) int smem[total_items];

  unsigned long long start = clock64();

  // Vectorized load from global memory to shared memory
#pragma unroll
  for (int i = 0; i < vectors_per_thread; ++i) {
    int4 vec = reinterpret_cast<const int4 *>(A)[tid + i * THREADS];
    reinterpret_cast<int4 *>(smem)[tid + i * THREADS] = vec;
  }

  __syncthreads(); // Ensure all threads have finished writing to shared memory

  // Vectorized store from shared memory to global memory
#pragma unroll
  for (int i = 0; i < vectors_per_thread; ++i) {
    int4 vec = reinterpret_cast<int4 *>(smem)[tid + i * THREADS];
    reinterpret_cast<int4 *>(B)[tid + i * THREADS] = vec;
  }

  unsigned long long end = clock64();
  times[tid] = end - start;
}

int main() {
  constexpr int ITEMS_PER_WARP = 8192;
  constexpr int BLOCKS = 1;
  constexpr int THREADS = 32;
  constexpr int n = BLOCKS * ITEMS_PER_WARP;

  int *A = static_cast<int *>(malloc(sizeof(int) * n));
  int *B = static_cast<int *>(malloc(sizeof(int) * n));

  // Initialize A with consecutive values starting from 0
  std::iota(A, A + n, 0);

  // Device pointers
  int *A_dev = nullptr;
  int *B_dev = nullptr;

  // Allocate device memory
  gpuErrchk(hipMalloc(&A_dev, n * sizeof(int)));
  gpuErrchk(hipMalloc(&B_dev, n * sizeof(int)));

  // Copy data from host to device
  gpuErrchk(hipMemcpy(A_dev, A, n * sizeof(int), hipMemcpyHostToDevice));

  // Create a device array to store the cycle count for each thread
  unsigned long long *times_dev = nullptr;
  gpuErrchk(
      hipMalloc(&times_dev, BLOCKS * THREADS * sizeof(unsigned long long)));

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start);

  // Launch kernel
  copy_and_measure<ITEMS_PER_WARP, THREADS>
      <<<BLOCKS, THREADS>>>(A_dev, B_dev, n, times_dev);

  hipEventRecord(stop);

  hipEventSynchronize(stop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);

  gpuErrchk(hipPeekAtLastError());

  // Copy back results
  gpuErrchk(hipMemcpy(B, B_dev, n * sizeof(int), hipMemcpyDeviceToHost));

  // Now copy the timing data from the device
  unsigned long long *times_host = new unsigned long long[BLOCKS * THREADS];
  gpuErrchk(hipMemcpy(times_host, times_dev,
                       BLOCKS * THREADS * sizeof(unsigned long long),
                       hipMemcpyDeviceToHost));

  // Print the per-thread cycle counts
  for (int i = 0; i < BLOCKS * THREADS; i++) {
    printf("Thread %d took %.1f cycles per element\n", i,
           static_cast<float>(times_host[i]) / n);
  }

  printf("Time taken: %f ms\n", milliseconds);

  for (int i = 0; i < n; i++) {
    assert(A[i] == B[i]);
  }

  // Cleanup
  delete[] times_host;
  hipFree(times_dev);
  hipFree(A_dev);
  hipFree(B_dev);
  free(A);
  free(B);

  return 0;
}