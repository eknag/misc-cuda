#include "hip/hip_runtime.h"
#include <assert.h>
#include <cstdint>
#include <cstdio>
#include <cuda/pipeline>
#include <cuda_pipeline_primitives.h>
#include <hip/hip_runtime.h>
#include <malloc.h>
#include <numeric>

#define gpuErrchk(ans)                                                         \
  {                                                                            \
    gpuAssert((ans), __FILE__, __LINE__);                                      \
  }
inline void gpuAssert(hipError_t code, const char *file, int line,
                      bool abort = true) {
  if (code != hipSuccess) {
    fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file,
            line);
    if (abort)
      exit(code);
  }
}

template <int TILE_SIZE, int THREADS>
__global__ void copy_and_measure(const int *__restrict__ A, int *__restrict__ B,
                                 int n, unsigned long long *times) {

  constexpr int tile_items_per_thread = TILE_SIZE / THREADS;
  constexpr int vector_size =
      4; // Using int4, which processes 4 integers at a time
  constexpr int tile_vectors_per_thread = tile_items_per_thread / vector_size;
  constexpr int tile_vectors = TILE_SIZE / vector_size;

  static_assert(tile_items_per_thread % (THREADS * vector_size) == 0,
                "ITEMS_PER_WARP must be divisible by THREADS * vector_size");

  assert(n % TILE_SIZE == 0);

  int tid = threadIdx.x;

  __shared__ __align__(16) int smem[TILE_SIZE];

  const int iters = n / TILE_SIZE;

  unsigned long long start = clock64();

  for (int iter = 0; iter < iters; ++iter) {

    // Vectorized load from global memory to shared memory
#pragma unroll
    for (int i = 0; i < tile_vectors_per_thread; ++i) {
      const int4 vec = reinterpret_cast<const int4 *>(
          A)[tid + i * THREADS + iter * tile_vectors];
      reinterpret_cast<int4 *>(smem)[tid + i * THREADS] = vec;
    }

#pragma unroll
    for (int i = 0; i < tile_vectors_per_thread; ++i) {
      const int4 vec = reinterpret_cast<int4 *>(smem)[tid + i * THREADS];
      reinterpret_cast<int4 *>(B)[tid + i * THREADS + iter * tile_vectors] =
          vec;
    }
  }
  __threadfence(); // Ensure all threads have finished writing to global memory
  unsigned long long end = clock64();
  times[tid] = end - start;
}

int main() {
  constexpr int TILE_SIZE = 8192;
  constexpr int ITEMS_PER_WARP = TILE_SIZE * 1024;
  constexpr int BLOCKS = 1;
  constexpr int THREADS = 32;
  constexpr int n = BLOCKS * ITEMS_PER_WARP;

  int *A = static_cast<int *>(malloc(sizeof(int) * n));
  int *B = static_cast<int *>(malloc(sizeof(int) * n));

  // Initialize A with consecutive values starting from 0
  std::iota(A, A + n, 0);

  // Device pointers
  int *A_dev = nullptr;
  int *B_dev = nullptr;

  // Allocate device memory
  gpuErrchk(hipMalloc(&A_dev, n * sizeof(int)));
  gpuErrchk(hipMalloc(&B_dev, n * sizeof(int)));

  // Copy data from host to device
  gpuErrchk(hipMemcpy(A_dev, A, n * sizeof(int), hipMemcpyHostToDevice));

  // Create a device array to store the cycle count for each thread
  unsigned long long *times_dev = nullptr;
  gpuErrchk(
      hipMalloc(&times_dev, BLOCKS * THREADS * sizeof(unsigned long long)));

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start);

  // Launch kernel
  copy_and_measure<TILE_SIZE, THREADS>
      <<<BLOCKS, THREADS>>>(A_dev, B_dev, n, times_dev);

  hipEventRecord(stop);

  hipEventSynchronize(stop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);

  gpuErrchk(hipPeekAtLastError());

  // Copy back results
  gpuErrchk(hipMemcpy(B, B_dev, n * sizeof(int), hipMemcpyDeviceToHost));

  // Now copy the timing data from the device
  unsigned long long *times_host = new unsigned long long[BLOCKS * THREADS];
  gpuErrchk(hipMemcpy(times_host, times_dev,
                       BLOCKS * THREADS * sizeof(unsigned long long),
                       hipMemcpyDeviceToHost));

  // Print the per-thread cycle counts
  unsigned long long max_cycles = 0;

  for (int i = 0; i < BLOCKS * THREADS; i++) {
    max_cycles = std::max(max_cycles, times_host[i]);
  }

  printf("%.1f cycles per element\n", static_cast<float>(max_cycles) / n);

  printf("Time taken: %f ms\n", milliseconds);

  for (int i = 0; i < n; i++) {
    assert(A[i] == B[i]);
  }

  // Cleanup
  delete[] times_host;
  hipFree(times_dev);
  hipFree(A_dev);
  hipFree(B_dev);
  free(A);
  free(B);

  return 0;
}