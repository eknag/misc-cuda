#include <cstdint>
#include <hip/hip_runtime.h>
#include <cstdio>

#define CHECK_CUDA(ans)                                                        \
  {                                                                            \
    gpuAssert((ans), __FILE__, __LINE__);                                      \
  }

inline void gpuAssert(hipError_t code, const char *file, int line,
                      bool abort = true) {
  if (code != hipSuccess) {
    fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file,
            line);
    if (abort)
      exit(code);
  }
}

struct CudaDeleter {
  void operator()(void *ptr) const { hipFree(ptr); }
};

template <typename T>
using cuda_unique_ptr = std::unique_ptr<T, CudaDeleter>;

template <typename T>
cuda_unique_ptr<T> make_cuda_unique(size_t n) {
  T *ptr = nullptr;
  CHECK_CUDA(hipMalloc(&ptr, n * sizeof(T)));
  return cuda_unique_ptr<T>(ptr);
}